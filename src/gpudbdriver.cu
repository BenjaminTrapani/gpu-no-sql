#include "hip/hip_runtime.h"
#include "gpudbdriver.hpp"
#include "Functors.hpp"

#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
#include "thrust/sort.h"
#include "thrust/copy.h"
#include "thrust/execution_policy.h"
#include "thrust/sequence.h"
#include "thrust/transform.h"
#include "thrust/device_ptr.h"
#include <unordered_map>

using namespace GPUDB;

GPUDBDriver::GPUDBDriver() {
    int nDevices;
    const unsigned int gb = 1024 * 1024 * 1024;
    hipGetDeviceCount(&nDevices);

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("CUDAGPUNoSQLDB starting...\n");
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  GPU Memory (GB): %f\n", ((float)prop.totalGlobalMem)/((float)(gb)));
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    }

    hipDeviceProp_t propOfInterest;
    hipGetDeviceProperties(&propOfInterest, 0);
    size_t memBytes = propOfInterest.totalGlobalMem;
    size_t allocSize = memBytes*0.12f; //0.12
    numEntries = allocSize/sizeof(Entry);
    printf("Num entries = %i\n", numEntries);

    //buffer allocation and initialization
    deviceEntries.reserve(numEntries);
    intermediateBuffer = new DeviceVector_t(numEntries);

    hostCreateBuffer = new HostVector_t();
    hostCreateBuffer->reserve(numEntries);

    hostResultBuffer = new HostVector_t();
    hostResultBuffer->reserve(numEntries);
}

GPUDBDriver::~GPUDBDriver() {
    delete intermediateBuffer;
    intermediateBuffer=0;

    delete hostCreateBuffer;
    hostCreateBuffer = 0;

    delete hostResultBuffer;
    hostResultBuffer = 0;
}

void GPUDBDriver::create(const Doc & toCreate) {
    create(toCreate.kvPair);
    for (std::list<Doc>::const_iterator iter = toCreate.children.begin(); iter != toCreate.children.end(); ++iter) {
        create(*iter);
    }
}

void GPUDBDriver::create(const Entry &object) {
    cpuAggregator.onEntryCreate(object);
    hostCreateBuffer->push_back(object);
}

void GPUDBDriver::batchCreate(std::vector<Doc> & docs) {
    for (std::vector<Doc>::iterator iter = docs.begin(); iter != docs.end(); ++iter){
        create(*iter);
    }
    syncCreates();
}

void GPUDBDriver::syncCreates() {
    DeviceVector_t::iterator oldEnd = deviceEntries.end();
    deviceEntries.resize(deviceEntries.size() + hostCreateBuffer->size());
    thrust::copy(hostCreateBuffer->begin(), hostCreateBuffer->end(), oldEnd);
    hostCreateBuffer->clear();
}

void GPUDBDriver::update(const Entry & searchFilter, const Entry & updates) {
    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(), ModifyEntry(updates),
                         MatchEntryByID(searchFilter.id));
    cpuAggregator.onUpdate(searchFilter.id, updates);
}

void GPUDBDriver::deleteBy(const Entry & searchFilter) {
    thrust::remove_if(deviceEntries.begin(), deviceEntries.end(), IsFullEntryMatch(searchFilter));
    cpuAggregator.onDelete(searchFilter.id);
}


void GPUDBDriver::optimizedSearchEntriesDown(const FilterGroup & filterGroup, const unsigned long int layer) {
    for (FilterGroup::const_iterator filterIter = filterGroup.group.begin(); filterIter != filterGroup.group.end();
         ++filterIter) {
        DeviceVector_t::iterator lastIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                            IsEntrySelected(layer));
        while (lastIter != deviceEntries.end()) {
            switch (filterIter->comparator){
                case GREATER:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryGreater>(IsEntryGreater(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case GREATER_EQ:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryGreaterEQ>(IsEntryGreaterEQ(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case EQ:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsPartialEntryMatch>(IsPartialEntryMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case LESS_EQ:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryLessEQ>(IsEntryLessEQ(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case LESS:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryLess>(IsEntryLess(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case KEY_ONLY:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<EntryKeyMatch>(EntryKeyMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case VAL_ONLY:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<EntryValMatch>(EntryValMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                default:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsPartialEntryMatch>(IsPartialEntryMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                }
            }

            lastIter = thrust::find_if(lastIter+1, deviceEntries.end(), IsEntrySelected(layer));
        }
    }
}

unsigned long int GPUDBDriver::internalGetDocsForFilterSet(const FilterSet &filters) {
    clock_t t1, t2;
    t1 = clock();
    thrust::transform(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(), UnselectEntry());

    for (FilterGroup::const_iterator firstGroupIter = filters[0].group.begin();
         firstGroupIter != filters[0].group.end();
         ++firstGroupIter) {
        switch (firstGroupIter->comparator){
            case GREATER:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     IsEntryGreater(firstGroupIter->entry));
                break;
            }
            case GREATER_EQ:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     IsEntryGreaterEQ(firstGroupIter->entry));
                break;
            }
            case EQ:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     IsPartialEntryMatch(firstGroupIter->entry));
                break;
            }
            case LESS_EQ:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     IsEntryLessEQ(firstGroupIter->entry));
                break;
            }
            case LESS:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     IsEntryLess(firstGroupIter->entry));
                break;
            }
            case KEY_ONLY:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     EntryKeyMatch(firstGroupIter->entry));
                break;
            }
            case VAL_ONLY:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntryTop(1, filters[0].resultMember),
                                     EntryValMatch(firstGroupIter->entry));
                break;
            }
            default:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                            SelectEntryTop(1, filters[0].resultMember),
                                            IsPartialEntryMatch(firstGroupIter->entry));
            }
        }
    }
    unsigned long long int layer = 1;
    for (FilterSet::const_iterator iter = filters.begin()+1; iter != filters.end(); ++iter) {
        optimizedSearchEntriesDown(*iter, layer);
        layer++;
    }
    t2 = clock();
    float diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("    Selecting all for query took %fms\n", diff);

    t1 = clock();
    markValidRootsForLayer(layer);
    t2 = clock();
    diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("    Marking valid roots took %fms\n", diff);
    return layer+1;
}

void GPUDBDriver::markValidRootsForLayer(const unsigned long long int beginLayer){
    DeviceVector_t::iterator iter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                    IsEntrySelected(beginLayer));
    while (iter != deviceEntries.end()){

        DeviceVector_t::iterator childIter = iter;
        Entry curHostChild = *childIter;

        if (curHostChild.isResultMember){
            thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                 SelectEntry(beginLayer+1, true),
                                 MatchEntryByID(curHostChild.id));
        }else {
            DeviceVector_t::iterator parentIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                                  GetElementWithChild(
                                                                          thrust::raw_pointer_cast(&(*childIter))));

            while (parentIter != deviceEntries.end()) {
                Entry hostParent = *parentIter;
                childIter = parentIter;

                if (hostParent.isResultMember) {
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(beginLayer + 1, true),
                                         MatchEntryByID(hostParent.id));
                    break;
                }
                parentIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                             GetElementWithChild(thrust::raw_pointer_cast(&(*parentIter))));
            }
        }
        iter = thrust::find_if(iter + 1, deviceEntries.end(),
                               IsEntrySelected(beginLayer));
    }
}

/*instead of building results using recursive find_if calls,
 *
 * 1. //won't work, flagging sub entries still requires sequence of find_if calls.
 * flag everything in the result set, copy_if to an intermediate buffer,
 * and copy back to host. Iterate through host vector. If an element's parent
 * id is a valid entry in hash table between id's and docs, add this element
 * as a child of the matched parent and create another table entry with the
 * pointer to the newly added entry. Otherwise, add the entry in the table. If an item
 * is marked as a result member, add it to the result vector of docs.
 *
 * //Will work but will take time to implement. Result buildup bottleneck will not be find_if but instead
 * //will be GPU-CPU transfer and CPU-side memory accesses.
 * 2. Keep a cpu-side table mapping id's to a list of child device Entry* pointers (populated on create and update in
 * constant time).
 * Lookup current parent->kvPair.id in the table, fetch data at each device address and add doc as subdoc. Do process as
 * before and build result tree top down recursively.
 * */
/*
 *
 *
 */

void GPUDBDriver::getDocumentsForParent(Doc * parent){
    clock_t t1, t2;
    t1 = clock();
    DeviceVector_t::iterator iter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                 GetElementWithParentID(parent->kvPair.id));
    t2 = clock();
    totalFindIfMs += ((float)(t2 - t1) / 1000000.0F ) * 1000;
    while (iter != deviceEntries.end()){
        size_t iterPos = thrust::distance(deviceEntries.begin(), iter);

        Doc * perm = parent->addChild(Doc(*iter));
        getDocumentsForParent(perm);
        t1 = clock();
        iter = thrust::find_if(iter+1, deviceEntries.end(),
                               GetElementWithParentID(parent->kvPair.id));
        t2 = clock();
        totalFindIfMs += ((float)(t2 - t1) / 1000000.0F ) * 1000;
    }
}
void GPUDBDriver::getDocumentsForRoots(const unsigned long int rootLayer, std::vector<Doc> & result){
    DeviceVector_t::iterator endPos = thrust::copy_if(deviceEntries.begin(), deviceEntries.end(), intermediateBuffer->begin(),
                    IsEntrySelected(rootLayer));

    size_t numCopied = thrust::distance(intermediateBuffer->begin(), endPos);

    thrust::copy(intermediateBuffer->begin(), endPos, hostResultBuffer->begin());
    cpuAggregator.buildResults(*hostResultBuffer, numCopied, result);
}

std::vector<Doc> GPUDBDriver::getDocumentsForFilterSet(const FilterSet & filters) {
    clock_t t1, t2;
    t1 = clock();
    unsigned long int finalLevel = internalGetDocsForFilterSet(filters);
    t2 = clock();
    float diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    //printf("  Select matches took %fms\n", diff);
    std::vector<Doc> result;
    t1 = clock();
    getDocumentsForRoots(finalLevel, result);
    t2 = clock();
    diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    //printf("  Build results took %fms\n", diff);
    return result;
}

unsigned long long int GPUDBDriver::getDocumentID(const FilterSet & sourceFilters) {
    std::vector<Doc> result = getDocumentsForFilterSet(sourceFilters);

    if (result.size() == 1) {
        return result[0].kvPair.id;
    }

    return 0;
}


