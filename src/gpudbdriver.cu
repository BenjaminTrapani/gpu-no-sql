#include "hip/hip_runtime.h"
#include "gpudbdriver.h"
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
#include "thrust/sort.h"
#include "thrust/copy.h"
#include "thrust/execution_policy.h"
#include "thrust/sequence.h"
#include "thrust/transform.h"
#include "thrust/device_ptr.h"

using namespace GPUDB;

GPUDBDriver::GPUDBDriver(){
    int nDevices;
    const unsigned int gb = 1024 * 1024 * 1024;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("CUDAGPUNoSQLDB starting...\n");
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  GPU Memory (GB): %f\n", ((float)prop.totalGlobalMem)/((float)(gb)));
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    }
    hipDeviceProp_t propOfInterest;
    hipGetDeviceProperties(&propOfInterest, 0);
    size_t memBytes = propOfInterest.totalGlobalMem;
    size_t allocSize = memBytes*0.05f;
    numEntries = allocSize/sizeof(CoreTupleType);
    printf("Num entries = %i\n", numEntries);

    //buffer allocation and initialization
    deviceEntries.reserve(numEntries);
    deviceIntermediateBuffer1 = new DeviceVector_t(numEntries);
    deviceIntermediateBuffer2 = new DeviceVector_t(numEntries);
    hostResultBuffer = new HostVector_t(numEntries);
}
GPUDBDriver::~GPUDBDriver(){
    delete deviceIntermediateBuffer1;
    deviceIntermediateBuffer1=0;

    delete deviceIntermediateBuffer2;
    deviceIntermediateBuffer2=0;

    delete hostResultBuffer;
    hostResultBuffer=0;
}

void GPUDBDriver::create(const CoreTupleType &object){
    deviceEntries.push_back(object);
}

struct IsPartialTupleMatch : thrust::unary_function<CoreTupleType,bool>{
    inline IsPartialTupleMatch(const CoreTupleType & filter):_filter(filter){}

    __device__ __host__
    inline bool operator()(const CoreTupleType & val)const{
        return val == _filter;
    }

private:
    const CoreTupleType _filter;
};

struct ExtractParentID : thrust::unary_function<CoreTupleType, GPUSizeType>{
    __device__ __host__
    inline GPUSizeType operator() (const CoreTupleType & val)const{
        return val.parentID;
    }
};

struct FetchTupleWithParentIDs : thrust::unary_function<CoreTupleType,bool>{
    inline FetchTupleWithParentIDs(CoreTupleType* validIndices,
                                   const size_t indexToExamine, const CoreTupleType & filter):
            _validIndices(validIndices), _indexToExamine(indexToExamine), _filter(filter){
    }

    __device__ __host__
    inline bool operator()(const CoreTupleType & ival)const{
        return _validIndices[_indexToExamine].parentID == ival.id && ival == _filter;
    }

private:
    CoreTupleType * _validIndices;
    const size_t _indexToExamine;
    const CoreTupleType _filter;
};

struct FetchDescendentTuple : thrust::unary_function<CoreTupleType, bool>{
    inline FetchDescendentTuple(const GPUSizeType desiredParentID): _desiredParentID(desiredParentID){}

    __device__ __host__
    inline bool operator()(const CoreTupleType & ival)const{
        return ival.parentID == _desiredParentID;
    }

private:
    GPUSizeType _desiredParentID;
};

void GPUDBDriver::update(const CoreTupleType &searchFilter, const CoreTupleType &updates){

}
void GPUDBDriver::deleteBy(const CoreTupleType &searchFilter){

}

void GPUDBDriver::sort(const CoreTupleType &sortFilter, const CoreTupleType &searchFilter){
}

void GPUDBDriver::searchEntries(const CoreTupleType & filter, DeviceVector_t * resultsFromThisStage,
                   DeviceVector_t * resultsFromLastStage,
                   const size_t numToSearch,
                   size_t &numFound){

    DeviceVector_t::iterator lastIter;
    numFound = 0;
    for(size_t i = 0; i < numToSearch; i++){
        lastIter = copy_if(deviceEntries.begin(), deviceEntries.begin() + numEntries,
                                resultsFromThisStage->begin() + numFound, FetchTupleWithParentIDs(
                                thrust::raw_pointer_cast(resultsFromLastStage->data()),
                                i,
                                filter));
        if(lastIter != resultsFromThisStage->end())
            numFound += thrust::distance(resultsFromThisStage->begin()+numFound, lastIter);
    }
}

QueryResult GPUDBDriver::getRootsForFilterSet(const std::vector<CoreTupleType> & filters){
    DeviceVector_t::iterator lastIter = copy_if(deviceEntries.begin(), deviceEntries.begin() + numEntries,
                                                deviceIntermediateBuffer1->begin(),
                                                IsPartialTupleMatch(filters[0]));
    size_t lastNumFound = thrust::distance(deviceIntermediateBuffer1->begin(), lastIter);
    size_t curNumFound = 0;

    DeviceVector_t * mostRecentResult = deviceIntermediateBuffer1;

    for(std::vector<CoreTupleType>::const_iterator iter = filters.begin()+1; iter != filters.end(); ++iter){
        size_t iterDistance = std::distance(filters.begin(), iter);
        if(iterDistance % 2 == 0){
            searchEntries(*iter, deviceIntermediateBuffer1, deviceIntermediateBuffer2, lastNumFound, curNumFound);
            mostRecentResult = deviceIntermediateBuffer1;
        }else{
            searchEntries(*iter, deviceIntermediateBuffer2, deviceIntermediateBuffer1, lastNumFound, curNumFound);
            mostRecentResult = deviceIntermediateBuffer2;
        }
        if(curNumFound==0) {
            break;
        }else{
            lastNumFound = curNumFound;
        }
    }

    QueryResult result;
    if(lastNumFound!=0) {
        printf("lastNumFound=%i\n", lastNumFound);
        *hostResultBuffer = *mostRecentResult;
        hostResultBuffer->resize(lastNumFound);
        result.numItems = lastNumFound;
        result.hostResultPointer = hostResultBuffer;
    }else{
        result.hostResultPointer = 0;
        result.numItems = 0;
    }
    return result;
}

QueryResult GPUDBDriver::getEntriesForRoots(const HostVector_t& roots, const size_t numRoots){
    DeviceVector_t::iterator lastIter;
    size_t numFound = 0;
    for(HostVector_t::const_iterator iter = roots.begin(); iter != roots.begin() + numRoots; ++iter){
        lastIter = thrust::copy_if(deviceEntries.begin(), deviceEntries.begin() + numEntries,
                           deviceIntermediateBuffer1->begin() + numFound, FetchDescendentTuple((*iter).id));
        numFound += thrust::distance(deviceIntermediateBuffer1->begin()+numFound, lastIter);
    }

    *hostResultBuffer = *deviceIntermediateBuffer1;
    for(size_t i = numFound; i < numFound + numRoots; i++){
        (*hostResultBuffer)[i] = roots[i-numFound];
    }

    QueryResult result;
    result.hostResultPointer = hostResultBuffer;
    result.numItems = numFound + numRoots;
    return result;
}

int main(int argc, char * argv[]){
    GPUDBDriver driver;
    printf("sizeof entry = %i\n", sizeof(Entry));

    for(unsigned int i = 0; i < driver.getTableSize()-2; i++){
        Entry anEntry;
        anEntry.data.bigVal=0;
        anEntry.valType = GPUDB_BGV;
        anEntry.key=i;
        anEntry.id = i;
        driver.create(anEntry);
    }
    Entry lastEntry;
    lastEntry.valType = GPUDB_BGV;
    lastEntry.data.bigVal = 1;
    lastEntry.key = 10;
    lastEntry.parentID = 3;
    driver.create(lastEntry);

    Entry realLastEntry;
    realLastEntry.valType = GPUDB_BGV;
    realLastEntry.id = 51;
    realLastEntry.data.bigVal = 1;
    realLastEntry.key = 10;
    realLastEntry.parentID = 6;
    driver.create(realLastEntry);

    Entry filter1 = realLastEntry;
    Entry filter2;
    filter2.data.bigVal=0;
    filter2.valType = GPUDB_BGV;
    filter2.key=realLastEntry.parentID;

    std::vector<Entry> filters;
    filters.push_back(filter1);
    filters.push_back(filter2);

    clock_t t1, t2;

    t1 = clock();
    QueryResult hostqueryResult = driver.getRootsForFilterSet(filters);
    t2 = clock();
    float diff1 = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("device multi-filter query latency = %fms\n", diff1);

    if(hostqueryResult.numItems) {
        for (HostVector_t::iterator iter = hostqueryResult.hostResultPointer->begin();
             iter != hostqueryResult.hostResultPointer->begin() + hostqueryResult.numItems;
                    ++iter){
            printf("Query result id = %llu\n", (*iter).id);
        }
    }

    t1 = clock();
    HostVector_t resultCopy = *hostqueryResult.hostResultPointer;
    QueryResult expandedRootResult = driver.getEntriesForRoots(resultCopy,
                                                               hostqueryResult.numItems);
    t2 = clock();
    float diff2 = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("device expand roots latency = %fms\n", diff2);

    if(expandedRootResult.numItems){
        for (HostVector_t::iterator iter = expandedRootResult.hostResultPointer->begin();
             iter != expandedRootResult.hostResultPointer->begin() + expandedRootResult.numItems;
             ++iter){
            printf("Expanded query result id = %llu\n", (*iter).id);
        }
    }

    return 0;

}