#include "hip/hip_runtime.h"
#include "gpudbdriver.hpp"
#include "Functors.hpp"

#include <hip/hip_runtime.h>
#include "stdio.h"
#include <time.h>
#include "thrust/sort.h"
#include "thrust/copy.h"
#include "thrust/execution_policy.h"
#include "thrust/sequence.h"
#include "thrust/transform.h"
#include "thrust/device_ptr.h"
#include <unordered_map>

using namespace GPUDB;

GPUDBDriver::GPUDBDriver() {
    int nDevices;
    const unsigned int gb = 1024 * 1024 * 1024;
    hipGetDeviceCount(&nDevices);

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("CUDAGPUNoSQLDB starting...\n");
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  GPU Memory (GB): %f\n", ((float)prop.totalGlobalMem)/((float)(gb)));
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    }

    hipDeviceProp_t propOfInterest;
    hipGetDeviceProperties(&propOfInterest, 0);
    size_t memBytes = propOfInterest.totalGlobalMem;
    size_t allocSize = memBytes*0.25f; //0.12
    numEntries = allocSize/sizeof(Entry);
    printf("Num entries = %i\n", numEntries);

    //buffer allocation and initialization
    deviceEntries.reserve(numEntries);

    //deviceIntermediateBuffer1 = new DeviceVector_t(numEntries);
    hostResultBuffer = new HostVector_t(numEntries);
    hostCreateBuffer = new HostVector_t();
    hostCreateBuffer->reserve(numEntries);
}

GPUDBDriver::~GPUDBDriver() {
    //delete deviceIntermediateBuffer1;
    //deviceIntermediateBuffer1=0;

    delete hostResultBuffer;
    hostResultBuffer=0;

    delete hostCreateBuffer;
    hostCreateBuffer = 0;
}

void GPUDBDriver::create(const Doc & toCreate) {
    create(toCreate.kvPair);
    for (std::list<Doc>::const_iterator iter = toCreate.children.begin(); iter != toCreate.children.end(); ++iter) {
        create(*iter);
    }
}

void GPUDBDriver::create(const Entry &object){
    hostCreateBuffer->push_back(object);
    //deviceEntries.push_back(object);
}

void GPUDBDriver::batchCreate(std::vector<Doc> & docs) {
    for (std::vector<Doc>::iterator iter = docs.begin(); iter != docs.end(); ++iter){
        create(*iter);
    }
    syncCreates();
}

void GPUDBDriver::syncCreates() {
    DeviceVector_t::iterator oldEnd = deviceEntries.end();
    deviceEntries.resize(deviceEntries.size() + hostCreateBuffer->size());
    thrust::copy(hostCreateBuffer->begin(), hostCreateBuffer->end(), oldEnd);
    hostCreateBuffer->clear();
}

void GPUDBDriver::update(const Entry & searchFilter, const Entry & updates) {
    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(), ModifyEntry(updates),
                         IsFullEntryMatch(searchFilter));
}

void GPUDBDriver::deleteBy(const Entry & searchFilter) {
    thrust::remove_if(deviceEntries.begin(), deviceEntries.end(), IsFullEntryMatch(searchFilter));
}


void GPUDBDriver::optimizedSearchEntriesDown(const FilterGroup & filterGroup, const unsigned long int layer) {
    for (FilterGroup::const_iterator filterIter = filterGroup.group.begin(); filterIter != filterGroup.group.end();
         ++filterIter) {
        DeviceVector_t::iterator lastIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                            IsEntrySelected(layer));
        while (lastIter != deviceEntries.end()) {
            switch (filterIter->comparator){
                case GREATER:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryGreater>(IsEntryGreater(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case GREATER_EQ:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryGreaterEQ>(IsEntryGreaterEQ(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case EQ:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsPartialEntryMatch>(IsPartialEntryMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case LESS_EQ:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryLessEQ>(IsEntryLessEQ(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case LESS:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsEntryLess>(IsEntryLess(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case KEY_ONLY:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<EntryKeyMatch>(EntryKeyMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                case VAL_ONLY:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<EntryValMatch>(EntryValMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                    break;
                }
                default:{
                    thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                         SelectEntry(layer+1, filterGroup.resultMember),
                                         FetchEntryWithChildID<IsPartialEntryMatch>(IsPartialEntryMatch(filterIter->entry),
                                                               thrust::raw_pointer_cast(&(*lastIter))
                                         ));
                }
            }

            lastIter = thrust::find_if(lastIter+1, deviceEntries.end(), IsEntrySelected(layer));
        }
    }
}

unsigned long int GPUDBDriver::selectAllSubelementsWithParentsSelected(const unsigned long int beginLayer){
    unsigned long int curLayer = beginLayer;
    bool foundOneOnLayer = true;

    while(foundOneOnLayer) {
        foundOneOnLayer = false;
        DeviceVector_t::iterator lastIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                            IsEntrySelected(curLayer));
        while (lastIter != deviceEntries.end()) {
            thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                 SelectEntry(curLayer + 1, false),
                                 GetElementWithParent(thrust::raw_pointer_cast(&(*lastIter))));
            lastIter = thrust::find_if(lastIter + 1, deviceEntries.end(),
                                       IsEntrySelected(curLayer));
            foundOneOnLayer = true;
        }
        if(foundOneOnLayer)
            curLayer++;
    }

    return curLayer-1;
}

unsigned long int GPUDBDriver::internalGetDocsForFilterSet(const FilterSet &filters) {
    clock_t t1, t2;
    t1 = clock();
    thrust::transform(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(), UnselectEntry());
    for (FilterGroup::const_iterator firstGroupIter = filters[0].group.begin();
         firstGroupIter != filters[0].group.end();
         ++firstGroupIter) {
        switch (firstGroupIter->comparator){
            case GREATER:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     IsEntryGreater(firstGroupIter->entry));
                break;
            }
            case GREATER_EQ:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     IsEntryGreaterEQ(firstGroupIter->entry));
                break;
            }
            case EQ:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     IsPartialEntryMatch(firstGroupIter->entry));
                break;
            }
            case LESS_EQ:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     IsEntryLessEQ(firstGroupIter->entry));
                break;
            }
            case LESS:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     IsEntryLess(firstGroupIter->entry));
                break;
            }
            case KEY_ONLY:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     EntryKeyMatch(firstGroupIter->entry));
                break;
            }
            case VAL_ONLY:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                     SelectEntry(1, filters[0].resultMember),
                                     EntryValMatch(firstGroupIter->entry));
                break;
            }
            default:{
                thrust::transform_if(deviceEntries.begin(), deviceEntries.end(), deviceEntries.begin(),
                                            SelectEntry(1, filters[0].resultMember),
                                            IsPartialEntryMatch(firstGroupIter->entry));
            }
        }
    }

    unsigned long int layer = 1;
    for (FilterSet::const_iterator iter = filters.begin()+1; iter != filters.end(); ++iter) {
        optimizedSearchEntriesDown(*iter, layer);
        layer++;
    }
    t2 = clock();
    float diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("    Selecting all for querry took %fms\n", diff);

    t1 = clock();
    layer = selectAllSubelementsWithParentsSelected(layer);
    t2 = clock();
    diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("    Selecting all subelements took %fms\n", diff);

    return layer;
}

void GPUDBDriver::buildResultsBottomUp(std::vector<Doc> & result, const unsigned long int beginLayer){
    /*DeviceVector_t::iterator lastIter = thrust::copy_if(deviceEntries.begin(), deviceEntries.end(),
                                                        deviceIntermediateBuffer1->begin(),
                                                        IsEntrySelected(beginLayer));*/

    std::unordered_map<unsigned long int, Doc> docIDMap;
    DeviceVector_t::iterator iter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                    IsEntrySelected(beginLayer));
    while (iter != deviceEntries.end()){

        DeviceVector_t::iterator childIter = iter;
        Entry curHostChild = *childIter;
        docIDMap[curHostChild.id] = Doc(curHostChild);
        DeviceVector_t::iterator parentIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                                              GetElementWithChild(thrust::raw_pointer_cast(&(*childIter))));

        Doc * lastValidParent;
        if (curHostChild.isResultMember) {
            lastValidParent = &docIDMap[curHostChild.id];
        } else {
            lastValidParent = 0;
        }
        while (parentIter != deviceEntries.end()) {
            Entry hostChild = *childIter;
            Entry hostParent = *parentIter;

            std::unordered_map<unsigned long int, Doc>::iterator keyIndex = docIDMap.find(hostParent.id);
            if(keyIndex == docIDMap.end()){
                docIDMap[hostParent.id] = Doc(hostParent);
            }

            docIDMap[hostParent.id].addChild(docIDMap[hostChild.id]);

            childIter = parentIter;

            if(hostParent.isResultMember){
                lastValidParent = &docIDMap[hostParent.id];
                break;
            }
            parentIter = thrust::find_if(deviceEntries.begin(), deviceEntries.end(),
                                         GetElementWithChild(thrust::raw_pointer_cast(&(*parentIter))));

        }
        if(lastValidParent) {
            result.push_back(*lastValidParent);
        }
        iter = thrust::find_if(iter + 1, deviceEntries.end(),
                               IsEntrySelected(beginLayer));
    }
}

std::vector<Doc> GPUDBDriver::getDocumentsForFilterSet(const FilterSet & filters) {
    clock_t t1, t2;
    t1 = clock();
    unsigned long int finalLevel = internalGetDocsForFilterSet(filters);
    t2 = clock();
    float diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("  Select matches took %fms\n", diff);
    std::vector<Doc> result;
    t1 = clock();
    buildResultsBottomUp(result, finalLevel);
    t2 = clock();
    diff = ((float)(t2 - t1) / 1000000.0F ) * 1000;
    printf("  Build results took %fms\n", diff);
    return result;
}

unsigned long long int GPUDBDriver::getDocumentID(const FilterSet & sourceFilters) {
    std::vector<Doc> result = getDocumentsForFilterSet(sourceFilters);

    if(result.size()==1)
        return getDocumentsForFilterSet(sourceFilters)[0].kvPair.id;

    return 0;
}


